
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <random>
#include <limits>
#include <cudnn.h>

#define CHECK(call)                                                  \
{                                                                    \
    const hipError_t error = call;                                  \
    std::cout << "CHECK cudaError_t: ";                              \
    if (error != hipSuccess)                                        \
    {                                                                \
        std::cout << __FILE__                                        \
                  << "("                                             \
                  << __LINE__                                        \
                  << ")"                                             \
                  << ": "                                            \
                  << "Error"                                         \
                  << std::endl;                                      \
        std::cout << "code: "                                        \
                  << error                                           \
                  << ", "                                            \
                  << "reason: "                                      \
                  << hipGetErrorString(error)                       \
                  << std::endl;                                      \
        exit(1);                                                     \
    }                                                                \
    else                                                             \
    {                                                                \
        std::cout << __FILE__                                        \
                  << "("                                             \
                  << __LINE__                                        \
                  << ")"                                             \
                  << ": "                                            \
                  << "hipSuccess"                                   \
                  << std::endl;                                      \
    }                                                                \
}

void rand_vector_float (std::vector<float> &v);

int main(int argc, char *argv[]) {
    cudnnHandle_t handle;
    cudnnCreate(&handle);

    int n = 5, c = 4, h = 1, w =1;

    std::vector<float> h_y(n * c * h* w, 0);
    std::vector<float> h_dy(n * c * h* w, 0);
    std::vector<float> h_dx(n * c * h* w, std::numeric_limits<float>::quiet_NaN());
    std::vector<float> h_dx_excpt(n * c * h* w, std::numeric_limits<float>::quiet_NaN());
    rand_vector_float(h_y);
    rand_vector_float(h_dy);

    for (std::vector<float>::const_iterator i = h_dy.begin(); i != h_dy.end(); ++i)
        std::cout << *i << ' ';
    std::cout << std::endl;

    cudnnTensorDescriptor_t yDesc;
    cudnnCreateTensorDescriptor(&yDesc);
    cudnnSetTensor4dDescriptor(yDesc,
                              CUDNN_TENSOR_NCHW,
                              CUDNN_DATA_FLOAT,
                              n,c,h,w);
    
    cudnnTensorDescriptor_t dyDesc;
    cudnnCreateTensorDescriptor(&dyDesc);
    cudnnSetTensor4dDescriptor(dyDesc,
                              CUDNN_TENSOR_NCHW,
                              CUDNN_DATA_FLOAT,
                              n,c,h,w);
    
    cudnnTensorDescriptor_t dxDesc;
    cudnnCreateTensorDescriptor(&dxDesc);
    cudnnSetTensor4dDescriptor(dxDesc,
                              CUDNN_TENSOR_NCHW,
                              CUDNN_DATA_FLOAT,
                              n,c,h,w);
    
    float *y, *dy, *dx;
    size_t size_ = n * c * h * w * sizeof(float);
    hipMalloc (&y, size_);
    hipMalloc (&dy, size_);
    hipMalloc (&dx, size_);
    hipMemset (dx, 0xff, size_);

    hipMemcpy(y, h_y.data(), size_, hipMemcpyHostToDevice);
    hipMemcpy(dy, h_dy.data(), size_, hipMemcpyHostToDevice);

    const float alpha = 1, beta = 0;
    cudnnSoftmaxBackward(
            handle,
            CUDNN_SOFTMAX_FAST,
            CUDNN_SOFTMAX_MODE_INSTANCE,
            &alpha,
            yDesc,
            y,
            dyDesc,
            dy,
            &beta,
            dxDesc,
            dx);

    CHECK(hipDeviceSynchronize());
    hipMemcpy(h_dx.data(), dx, size_, hipMemcpyDeviceToHost);

    for (std::vector<float>::const_iterator i = h_dx.begin(); i != h_dx.end(); ++i)
        std::cout << *i << ' ';
    std::cout << std::endl;

    hipFree(y);
    hipFree(dy);
    hipFree(dx);
    cudnnDestroyTensorDescriptor(yDesc);
    cudnnDestroyTensorDescriptor(dyDesc);
    cudnnDestroyTensorDescriptor(dxDesc);
    cudnnDestroy(handle);
    CHECK(hipDeviceSynchronize());
    return 0;
}

void pseudosoftmaxbackward() {
    return;
}

std::mt19937 mt(0);
void rand_vector_float (std::vector<float> &v) {
    std::normal_distribution<> rand(0, 5);
    for (std::vector<float>::iterator i = v.begin(); i != v.end(); ++i) {
        *i = rand(mt);
    }
    return;
}
