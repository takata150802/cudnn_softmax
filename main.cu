
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <random>
#include <limits>
#include <cudnn.h>

#define CHECK(call)                                                  \
{                                                                    \
    const hipError_t error = call;                                  \
    std::cout << "CHECK cudaError_t: ";                              \
    if (error != hipSuccess)                                        \
    {                                                                \
        std::cout << __FILE__                                        \
                  << "("                                             \
                  << __LINE__                                        \
                  << ")"                                             \
                  << ": "                                            \
                  << "Error"                                         \
                  << std::endl;                                      \
        std::cout << "code: "                                        \
                  << error                                           \
                  << ", "                                            \
                  << "reason: "                                      \
                  << hipGetErrorString(error)                       \
                  << std::endl;                                      \
        exit(1);                                                     \
    }                                                                \
    else                                                             \
    {                                                                \
        std::cout << __FILE__                                        \
                  << "("                                             \
                  << __LINE__                                        \
                  << ")"                                             \
                  << ": "                                            \
                  << "hipSuccess"                                   \
                  << std::endl;                                      \
    }                                                                \
}

void rand_vector_float (std::vector<float> &v);
void pseudoSoftmaxBackward(const std::vector<float> &y, 
                           const std::vector<float> &dy,
                           std::vector<float> &dx,
                           const int N, const int  C, const int H, const int W);

int main(int argc, char *argv[]) {
    cudnnHandle_t handle;
    cudnnCreate(&handle);

    int n = 5, c = 4, h = 1, w =1;

    std::vector<float> h_y(n * c * h* w, 0);
    std::vector<float> h_dy(n * c * h* w, 0);
    std::vector<float> h_dx(n * c * h* w, std::numeric_limits<float>::quiet_NaN());
    std::vector<float> h_dx_expct(n * c * h* w, std::numeric_limits<float>::quiet_NaN());
    rand_vector_float(h_y);
    rand_vector_float(h_dy);

    cudnnTensorDescriptor_t yDesc;
    cudnnCreateTensorDescriptor(&yDesc);
    cudnnSetTensor4dDescriptor(yDesc,
                              CUDNN_TENSOR_NCHW,
                              CUDNN_DATA_FLOAT,
                              n,c,h,w);
    
    cudnnTensorDescriptor_t dyDesc;
    cudnnCreateTensorDescriptor(&dyDesc);
    cudnnSetTensor4dDescriptor(dyDesc,
                              CUDNN_TENSOR_NCHW,
                              CUDNN_DATA_FLOAT,
                              n,c,h,w);
    
    cudnnTensorDescriptor_t dxDesc;
    cudnnCreateTensorDescriptor(&dxDesc);
    cudnnSetTensor4dDescriptor(dxDesc,
                              CUDNN_TENSOR_NCHW,
                              CUDNN_DATA_FLOAT,
                              n,c,h,w);
    
    float *y, *dy, *dx;
    size_t size_ = n * c * h * w * sizeof(float);
    hipMalloc (&y, size_);
    hipMalloc (&dy, size_);
    hipMalloc (&dx, size_);
    hipMemset (dx, 0xff, size_);

    hipMemcpy(y, h_y.data(), size_, hipMemcpyHostToDevice);
    hipMemcpy(dy, h_dy.data(), size_, hipMemcpyHostToDevice);

    const float alpha = 1, beta = 0;
    cudnnSoftmaxBackward(
            handle,
            CUDNN_SOFTMAX_FAST,
            CUDNN_SOFTMAX_MODE_CHANNEL,
            &alpha,
            yDesc,
            y,
            dyDesc,
            dy,
            &beta,
            dxDesc,
            dx);

    pseudoSoftmaxBackward(h_y, h_dy, h_dx_expct, n, c, h, w);

    CHECK(hipDeviceSynchronize());
    hipMemcpy(h_dx.data(), dx, size_, hipMemcpyDeviceToHost);

    for (std::vector<float>::const_iterator i = h_dx.begin(); i != h_dx.end(); ++i)
        std::cout << *i << ' ';
    std::cout << std::endl;

    for (std::vector<float>::const_iterator i = h_dx_expct.begin(); i != h_dx_expct.end(); ++i)
        std::cout << *i << ' ';
    std::cout << std::endl;

    hipFree(y);
    hipFree(dy);
    hipFree(dx);
    cudnnDestroyTensorDescriptor(yDesc);
    cudnnDestroyTensorDescriptor(dyDesc);
    cudnnDestroyTensorDescriptor(dxDesc);
    cudnnDestroy(handle);
    CHECK(hipDeviceSynchronize());
    return 0;
}

namespace {
    int getIndex(const int n,
                 const int c,
                 const int h,
                 const int w,
                 const int N,
                 const int C,
                 const int H,
                 const int W
                 ) {
        return n * C * H * W
                 + c * H * W
                     + h * W
                         + w;
    }
}

void pseudoSoftmaxBackward(const std::vector<float> &y, 
                           const std::vector<float> &dy,
                           std::vector<float> &dx,
                           const int N, const int  C, const int H, const int W) {
    int idx, idx_s;
    std::vector<float> sum_(N * 1 * H * W, 0);

    for (int n = 0; n < N; ++n) {
        for (int c = 0; c < C; ++c) {
            for (int h = 0; h < H; ++h) {
                for (int w = 0; w < W; ++w) {
                    idx = getIndex(n, c, h, w, N, C, H, W);
                    idx_s = getIndex(n, 1, h, w, N, 1, H, W);
                    sum_[idx_s] += y[idx] * dy[idx];
                }
            }
        }
    }
    for (int n = 0; n < N; ++n) {
        for (int c = 0; c < C; ++c) {
            for (int h = 0; h < H; ++h) {
                for (int w = 0; w < W; ++w) {
                    idx = getIndex(n, c, h, w, N, C, H, W);
                    idx_s = getIndex(n, 1, h, w, N, 1, H, W);
                    dx[idx] = y[idx] * (dy[idx] - sum_[idx_s]);
                }
            }
        }
    }
    return;
}

std::mt19937 mt(0);
void rand_vector_float (std::vector<float> &v) {
    std::normal_distribution<> rand(0, 5);
    for (std::vector<float>::iterator i = v.begin(); i != v.end(); ++i) {
        *i = rand(mt);
    }
    return;
}
