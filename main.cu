
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <cudnn.h>

#define CHECK(call)                                                  \
{                                                                    \
    const hipError_t error = call;                                  \
    std::cout << "CHECK cudaError_t: ";                              \
    if (error != hipSuccess)                                        \
    {                                                                \
        std::cout << __FILE__                                        \
                  << "("                                             \
                  << __LINE__                                        \
                  << ")"                                             \
                  << ": "                                            \
                  << "Error"                                         \
                  << std::endl;                                      \
        std::cout << "code: "                                        \
                  << error                                           \
                  << ", "                                            \
                  << "reason: "                                      \
                  << hipGetErrorString(error)                       \
                  << std::endl;                                      \
        exit(1);                                                     \
    }                                                                \
    else                                                             \
    {                                                                \
        std::cout << __FILE__                                        \
                  << "("                                             \
                  << __LINE__                                        \
                  << ")"                                             \
                  << ": "                                            \
                  << "hipSuccess"                                   \
                  << std::endl;                                      \
    }                                                                \
}

int main(int argc, char *argv[]) {
    cudnnHandle_t handle;
    cudnnCreate(&handle);

    int n = 5, c = 4, h = 1, w =1;

    std::vector<float> h_y(n * c * h* w, 0);
    std::vector<float> h_dy(n * c * h* w, 1);
    std::vector<float> h_dx(n * c * h* w, -1);

    cudnnTensorDescriptor_t yDesc;
    cudnnCreateTensorDescriptor(&yDesc);
    cudnnSetTensor4dDescriptor(yDesc,
                              CUDNN_TENSOR_NCHW,
                              CUDNN_DATA_FLOAT,
                              n,c,h,w);
    
    cudnnTensorDescriptor_t dyDesc;
    cudnnCreateTensorDescriptor(&dyDesc);
    cudnnSetTensor4dDescriptor(dyDesc,
                              CUDNN_TENSOR_NCHW,
                              CUDNN_DATA_FLOAT,
                              n,c,h,w);
    
    cudnnTensorDescriptor_t dxDesc;
    cudnnCreateTensorDescriptor(&dxDesc);
    cudnnSetTensor4dDescriptor(dxDesc,
                              CUDNN_TENSOR_NCHW,
                              CUDNN_DATA_FLOAT,
                              n,c,h,w);
    
    float *y, *dy, *dx;
    size_t size_ = n * c * h * w * sizeof(float);
    hipMalloc (&y, size_);
    hipMalloc (&dy, size_);
    hipMalloc (&dx, size_);
    hipMemset (dx, 0xff, size_);

    hipMemcpy(y, h_y.data(), size_, hipMemcpyHostToDevice);
    hipMemcpy(dy, h_dy.data(), size_, hipMemcpyHostToDevice);

    const float alpha = 1, beta = 0;
    cudnnSoftmaxBackward(
            handle,
            CUDNN_SOFTMAX_FAST,
            CUDNN_SOFTMAX_MODE_INSTANCE,
            &alpha,
            yDesc,
            y,
            dyDesc,
            dy,
            &beta,
            dxDesc,
            dx);

    hipMemcpy(h_dx.data(), dx, size_, hipMemcpyDeviceToHost);

    for (std::vector<float>::const_iterator i = h_dx.begin(); i != h_dx.end(); ++i)
        std::cout << *i << ' ';
    std::cout << std::endl;

    hipFree(y);
    hipFree(dy);
    hipFree(dx);
    cudnnDestroyTensorDescriptor(yDesc);
    cudnnDestroyTensorDescriptor(dyDesc);
    cudnnDestroyTensorDescriptor(dxDesc);
    cudnnDestroy(handle);
    CHECK(hipDeviceSynchronize());
    return 0;
}
